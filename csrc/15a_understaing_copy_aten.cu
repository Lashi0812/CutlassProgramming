#include "hip/hip_runtime.h"
#include "cute/swizzle.hpp"
#include "cute/tensor.hpp"
#include "cute/algorithm/functional.hpp"
#include "cute/algorithm/tensor_algorithms.hpp"
#include "cute/layout.hpp"
#include "cute/numeric/int.hpp"
#include "cute/pointer.hpp"
#include "cute/arch/copy_sm80.hpp"
#include "cute/numeric/half.hpp"
#include "cute/atom/copy_atom.hpp"
#include "cute/algorithm/copy.hpp"
#include <ATen/ATen.h>
#include <iostream>
#include <numeric>


using namespace cute;

///////////////////////////////////////////////////////////////////////////////////////////////
//                      Generic Copy using LD
///////////////////////////////////////////////////////////////////////////////////////////////


template <typename TiledCopy, typename T>
__global__ void copy_kernel(T const *in, T *out) {

    TiledCopy tiled_copy;
    __shared__ T smemA[cosize_v<Layout<typename TiledCopy::TiledShape_MN>>];

    auto swizzle_layout = composition(Swizzle<2, 3, 3>{}, Layout<Shape<_16, _16>>{});
    // auto smem_layout = tile_to_shape(swizzle_layout, Shape<_16, _16>{});
    auto gA = make_tensor(make_gmem_ptr(in), typename TiledCopy::TiledShape_MN{});
    // auto sA = make_tensor(make_smem_ptr(smemA), typename TiledCopy::TiledShape_MN{});

    auto sA = make_tensor(make_smem_ptr(smemA), swizzle_layout);
    auto gC = make_tensor(make_gmem_ptr(out), typename TiledCopy::TiledShape_MN{});

    auto thr_copy = tiled_copy.get_slice(threadIdx.x);
    auto tAgA = thr_copy.partition_S(gA);
    auto tAsA = thr_copy.partition_D(sA);
    auto tAgC = thr_copy.partition_S(gC);

    auto tArA = make_fragment_like(tAsA);

    copy(tiled_copy, tAgA, tAsA);
    cp_async_fence();
    cp_async_wait<0>();

    // print_helper(tAgA, tAsA);

    // clang-format off
        // print(gA);print("\n");
        // print(sA);print("\n");
        // print(tAgA);print("\n");
        // print(tAsA);print("\n");
        // print("Common Vector GS : ");print(max_common_vector(tAgA, tAsA));print("\n");
        // print("Common Vector SR : ");print(max_common_vector(tAsA, tArA));print("\n");
        // print("Common Vector RG : ");print(max_common_vector(tArA, tAgC));print("\n");
    // clang-format on

    // for (int i{0}; i < size(tAgA); ++i)
    //     tAsA(i) = tAgA(i);
    // copy(Copy_Atom<UniversalCopy<uint32_t>,half_t>{},tAgA, tAsA);

    copy(tAsA, tArA);
    // for(int i{0};i<size(tAsA);++i)
    //   tArA(i) = tAsA(i);

    transform(tArA, pre_increment{});

    copy(tArA, tAgC);
}

void test_copy_host() {

    // using TiledCopy = decltype(make_tiled_copy(
    //   Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, half_t>{},
    //   Layout<Shape<_2, _16>>{},
    //   Layout<Shape<_8, _1>>{}));

    using TiledCopy = decltype(make_tiled_copy(
      Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<cute::uint32_t>, half_t>{},
      Layout<Shape<_2, _16>>{},
      Layout<Shape<_8, _1>>{}));

    TiledCopy tiled_copy;
    std::vector<int> vecG(256);
    std::vector<int> vecS(256);
    std::iota(vecG.begin(), vecG.end(), 0);
    std::iota(vecS.begin(), vecS.end(), 0);

    auto swizzle_layout = composition(Swizzle<3, 1, 5>{}, Layout<Shape<_16, _16>>{});
    // auto smem_layout = tile_to_shape(swizzle_layout, Shape<_16, _16>{});
    // auto gA = make_counting_tensor(Layout<typename TiledCopy::TiledShape_MN>{});
    // auto sA = make_counting_tensor(Layout<typename TiledCopy::TiledShape_MN>{});
    auto gA = make_tensor(vecG.data(), Layout<typename TiledCopy::TiledShape_MN>{});
    auto sA = make_tensor(vecS.data(), swizzle_layout);

    auto thr_copy = tiled_copy.get_slice(8);
    auto tAgA = thr_copy.partition_S(gA);
    auto tAsA = thr_copy.partition_D(sA);

    // copy(tAgA, tAsA);

    // clang-format off
    print(gA);print("\n");
    print(sA);print("\n");print_tensor(sA);    
    print(tAgA);print("\n");print_tensor(tAgA);
    print(tAsA);print("\n");print_tensor(tAsA);
    print("Common Vector : ");print(max_common_vector(tAgA, tAsA));print("\n");
    // clang-format on
}

void test_normal_copy() {
    using tiled_copy = decltype(make_tiled_copy(
      Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, half_t>{},
      Layout<Shape<_2, _16>>{},
      Layout<Shape<_8, _1>>{}));

    auto in =
      at::arange(
        decltype(size(tiled_copy::TiledShape_MN{}))::value, at::TensorOptions().dtype(at::kHalf))
        .reshape({size<0>(tiled_copy::TiledShape_MN{}), size<1>(tiled_copy::TiledShape_MN{})});

    auto out = at::zeros_like(in);

    half_t *d_in, *d_out;
    hipMalloc((void **)&d_in, in.numel() * in.element_size());
    hipMalloc((void **)&d_out, out.numel() * out.element_size());

    hipMemcpy(d_in, in.data_ptr(), in.numel() * in.element_size(), hipMemcpyHostToDevice);
    hipMemcpy(d_out, out.data_ptr(), out.numel() * out.element_size(), hipMemcpyHostToDevice);

    copy_kernel<tiled_copy><<<1, 32>>>(d_in, d_out);
    hipMemcpy(out.data_ptr(), d_out, out.numel() * out.element_size(), hipMemcpyDeviceToHost);
    std::cout << out << std::endl;

    hipFree(d_in);
    hipFree(d_out);
}

int main() {
    test_normal_copy();
    hipDeviceReset();

    // test_copy_host();
}