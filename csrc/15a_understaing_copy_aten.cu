#include "hip/hip_runtime.h"
#include "cute/stride.hpp"
#include "cute/tensor.hpp"
#include "cute/algorithm/functional.hpp"
#include "cute/algorithm/tensor_algorithms.hpp"
#include "cute/layout.hpp"
#include "cute/numeric/int.hpp"
#include "cute/pointer.hpp"
#include "cute/arch/copy_sm80.hpp"
#include "cute/numeric/half.hpp"
#include "cute/atom/copy_atom.hpp"
#include "cute/algorithm/copy.hpp"
#include <ATen/ATen.h>
#include <iostream>
#include <vector>
#include "cute/util/debug.hpp"
#include "latex.hpp"

using namespace cute;

template <typename TiledCopy, typename T>
__global__ void copy_kernel(T const *in, T *out) {

    TiledCopy tiled_copy;
    __shared__ T smemA[cosize_v<Layout<typename TiledCopy::TiledShape_MN>>];

    auto gA = make_tensor(make_gmem_ptr(in), typename TiledCopy::TiledShape_MN{});
    auto sA = make_tensor(make_smem_ptr(smemA), typename TiledCopy::TiledShape_MN{});
    auto gC = make_tensor(make_gmem_ptr(out), typename TiledCopy::TiledShape_MN{});

    auto thr_copy = tiled_copy.get_slice(threadIdx.x);
    auto tAgA = thr_copy.partition_S(gA);
    auto tAsA = thr_copy.partition_D(sA);
    auto tAgC = thr_copy.partition_S(gC);

    auto tArA = make_fragment_like(tAsA);

    // if (thread0()) {
    //     // clang-format off
    //     print(gA);print("\n");
    //     print(sA);print("\n");
    //     print(tAgA);print("\n");
    //     print(tAsA);print("\n");
    //     print("Common Vector : ");print(max_common_vector(tAgA, tAsA));print("\n");
    //     // clang-format on
    // }
    copy(tAgA, tAsA);
    cp_async_fence();
    cp_async_wait<0>();
    
    copy(tAsA, tArA);

    transform(tArA, pre_increment{});

    copy(tArA, tAgC);
}

void test_copy_host() {

    // using TiledCopy = decltype(make_tiled_copy(
    //   Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, half_t>{},
    //   Layout<Shape<_2, _16>>{},
    //   Layout<Shape<_8, _1>>{}));

    using TiledCopy = decltype(
    make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<cute::uint128_t>, half_t>{},
                    Layout<Shape <_16,_8>,
                           Stride< _8,_1>>{},
                    Layout<Shape < _1,_8>>{}));

    TiledCopy tiled_copy;
    std::vector<int> vecG(1024);
    std::vector<int> vecS(1024);
    // auto gA = make_counting_tensor(Layout<typename TiledCopy::TiledShape_MN>{});
    // auto sA = make_counting_tensor(Layout<typename TiledCopy::TiledShape_MN>{});
    auto gA = make_tensor(vecG.data(), Layout<typename TiledCopy::TiledShape_MN,Stride<_64,_1>>{});
    auto sA = make_tensor(vecS.data(), Layout<typename TiledCopy::TiledShape_MN,Stride<_64,_1>>{});

    auto thr_copy = tiled_copy.get_slice(0);
    auto tAgA = thr_copy.partition_S(gA);
    auto tAsA = thr_copy.partition_D(sA);

    // clang-format off
    print(gA);print("\n");
    print(sA);print("\n");
    print(tAgA);print("\n");
    print(tAsA);print("\n");
    print("Common Vector : ");print(max_common_vector(tAgA, tAsA));print("\n");
    // clang-format on

    copy(tAgA, tAsA);
}

void test_normal_copy() {
    using tiled_copy = decltype(make_tiled_copy(
      Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, half_t>{},
      Layout<Shape<_2, _16>>{},
      Layout<Shape<_8, _1>>{}));

    auto in =
      at::arange(
        decltype(size(tiled_copy::TiledShape_MN{}))::value, at::TensorOptions().dtype(at::kHalf))
        .reshape({size<0>(tiled_copy::TiledShape_MN{}), size<1>(tiled_copy::TiledShape_MN{})});

    auto out = at::zeros_like(in);

    half_t *d_in, *d_out;
    hipMalloc((void **)&d_in, in.numel() * in.element_size());
    hipMalloc((void **)&d_out, out.numel() * out.element_size());

    hipMemcpy(d_in, in.data_ptr(), in.numel() * in.element_size(), hipMemcpyHostToDevice);
    hipMemcpy(d_out, out.data_ptr(), out.numel() * out.element_size(), hipMemcpyHostToDevice);

    copy_kernel<tiled_copy><<<1, 32>>>(d_in, d_out);
    hipMemcpy(out.data_ptr(), d_out, out.numel() * out.element_size(), hipMemcpyDeviceToHost);
    std::cout << out << std::endl;

    hipFree(d_in);
    hipFree(d_out);

    // print_latex(tiled_copy{}, "CP_T16x2_V1x8", 3);
}

int main() {
    test_normal_copy();
    hipDeviceReset();
    // test_copy_host();
}