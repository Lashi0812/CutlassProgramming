#include "hip/hip_runtime.h"
#include "cute/tensor.hpp"
#include "cute/arch/copy_sm75.hpp"
#include "cute/arch/mma_sm80.hpp"
#include "cute/atom/mma_atom.hpp"
#include "cute/swizzle.hpp"
#include "cute/algorithm/functional.hpp"
#include "cute/algorithm/tensor_algorithms.hpp"
#include "cute/layout.hpp"
#include "cute/numeric/int.hpp"
#include "cute/pointer.hpp"
#include "cute/arch/copy_sm80.hpp"
#include "cute/numeric/half.hpp"
#include "cute/atom/copy_atom.hpp"
#include "cute/algorithm/copy.hpp"
#include "cute/util/print.hpp"
#include "latex.hpp"
#include <ATen/ATen.h>
#include <ATen/ops/rand.h>
#include <ATen/ops/zeros_like.h>
#include <c10/core/ScalarType.h>
#include <cstdint>
#include <iostream>
#include <numeric>

using namespace cute;

///////////////////////////////////////////////////////////////////////////////////////////////
//                      Generic Copy using LD
///////////////////////////////////////////////////////////////////////////////////////////////

template <typename TiledCopy, typename T>
__global__ void copy_kernel(T const *in, T *out) {

    TiledCopy tiled_copy;
    __shared__ T smemA[cosize_v<Layout<typename TiledCopy::TiledShape_MN>>];

    auto swizzle_layout = composition(Swizzle<2, 3, 3>{}, Layout<Shape<_16, _16>>{});
    // auto smem_layout = tile_to_shape(swizzle_layout, Shape<_16, _16>{});
    auto gA = make_tensor(make_gmem_ptr(in), typename TiledCopy::TiledShape_MN{});
    // auto sA = make_tensor(make_smem_ptr(smemA), typename TiledCopy::TiledShape_MN{});

    auto sA = make_tensor(make_smem_ptr(smemA), swizzle_layout);
    auto gC = make_tensor(make_gmem_ptr(out), typename TiledCopy::TiledShape_MN{});

    auto thr_copy = tiled_copy.get_slice(threadIdx.x);
    auto tAgA = thr_copy.partition_S(gA);
    auto tAsA = thr_copy.partition_D(sA);
    auto tAgC = thr_copy.partition_S(gC);

    auto tArA = make_fragment_like(tAsA);

    copy(tiled_copy, tAgA, tAsA);
    cp_async_fence();
    cp_async_wait<0>();

    // print_helper(tAgA, tAsA);

    // clang-format off
        // print(gA);print("\n");
        // print(sA);print("\n");
        // print(tAgA);print("\n");
        // print(tAsA);print("\n");
        // print("Common Vector GS : ");print(max_common_vector(tAgA, tAsA));print("\n");
        // print("Common Vector SR : ");print(max_common_vector(tAsA, tArA));print("\n");
        // print("Common Vector RG : ");print(max_common_vector(tArA, tAgC));print("\n");
    // clang-format on

    // for (int i{0}; i < size(tAgA); ++i)
    //     tAsA(i) = tAgA(i);
    // copy(Copy_Atom<UniversalCopy<uint32_t>,half_t>{},tAgA, tAsA);

    copy(tAsA, tArA);
    // for(int i{0};i<size(tAsA);++i)
    //   tArA(i) = tAsA(i);

    transform(tArA, pre_increment{});

    copy(tArA, tAgC);
}

void test_copy_host() {

    // using TiledCopy = decltype(make_tiled_copy(
    //   Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, half_t>{},
    //   Layout<Shape<_2, _16>>{},
    //   Layout<Shape<_8, _1>>{}));

    using TiledCopy = decltype(make_tiled_copy(
      Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<cute::uint32_t>, half_t>{},
      Layout<Shape<_2, _16>>{},
      Layout<Shape<_8, _1>>{}));

    TiledCopy tiled_copy;
    std::vector<int> vecG(256);
    std::vector<int> vecS(256);
    std::iota(vecG.begin(), vecG.end(), 0);
    std::iota(vecS.begin(), vecS.end(), 0);

    auto swizzle_layout = composition(Swizzle<3, 1, 5>{}, Layout<Shape<_16, _16>>{});
    // auto smem_layout = tile_to_shape(swizzle_layout, Shape<_16, _16>{});
    // auto gA = make_counting_tensor(Layout<typename TiledCopy::TiledShape_MN>{});
    // auto sA = make_counting_tensor(Layout<typename TiledCopy::TiledShape_MN>{});
    auto gA = make_tensor(vecG.data(), Layout<typename TiledCopy::TiledShape_MN>{});
    auto sA = make_tensor(vecS.data(), swizzle_layout);

    auto thr_copy = tiled_copy.get_slice(8);
    auto tAgA = thr_copy.partition_S(gA);
    auto tAsA = thr_copy.partition_D(sA);

    // copy(tAgA, tAsA);

    // clang-format off
    print(gA);print("\n");
    print(sA);print("\n");print_tensor(sA);    
    print(tAgA);print("\n");print_tensor(tAgA);
    print(tAsA);print("\n");print_tensor(tAsA);
    print("Common Vector : ");print(max_common_vector(tAgA, tAsA));print("\n");
    // clang-format on
}

void test_normal_copy() {
    using tiled_copy = decltype(make_tiled_copy(
      Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, half_t>{},
      Layout<Shape<_2, _16>>{},
      Layout<Shape<_8, _1>>{}));

    auto in =
      at::arange(
        decltype(size(tiled_copy::TiledShape_MN{}))::value, at::TensorOptions().dtype(at::kHalf))
        .reshape({size<0>(tiled_copy::TiledShape_MN{}), size<1>(tiled_copy::TiledShape_MN{})});

    auto out = at::zeros_like(in);

    half_t *d_in, *d_out;
    hipMalloc((void **)&d_in, in.numel() * in.element_size());
    hipMalloc((void **)&d_out, out.numel() * out.element_size());

    hipMemcpy(d_in, in.data_ptr(), in.numel() * in.element_size(), hipMemcpyHostToDevice);
    hipMemcpy(d_out, out.data_ptr(), out.numel() * out.element_size(), hipMemcpyHostToDevice);

    copy_kernel<tiled_copy><<<1, 32>>>(d_in, d_out);
    hipMemcpy(out.data_ptr(), d_out, out.numel() * out.element_size(), hipMemcpyDeviceToHost);
    std::cout << out << std::endl;

    hipFree(d_in);
    hipFree(d_out);
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                                          ldmatrix copy
// * Moving the data from shared memory to Register.
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename GmemTiledCopy, typename SmemCopyAtom, typename SmemLayout, typename T>
__global__ void matrix_copy_kernel(T const *in, T *out) {
    __shared__ T smem[cosize_v<SmemLayout>];

    GmemTiledCopy tiled_copy;
    auto gA = make_tensor(make_gmem_ptr(in), SmemLayout{});
    auto sA = make_tensor(make_smem_ptr(smem), SmemLayout{});
    auto gC = make_tensor(make_gmem_ptr(out), SmemLayout{});

    auto thr_copy = tiled_copy.get_thread_slice(threadIdx.x);
    auto tAgA = thr_copy.partition_S(gA);
    auto tAsA = thr_copy.partition_D(sA);

    copy(tiled_copy, tAgA, tAsA);
    cp_async_fence();
    cp_async_wait<0>();

    SmemCopyAtom smem_copy_atom;
    auto smem_thr_copy = smem_copy_atom.get_thread_slice(threadIdx.x);
    auto tCsA = smem_thr_copy.partition_S(sA);
    auto tCgc = smem_thr_copy.partition_D(gC);
    auto tCrA = make_tensor<T>(shape(tCgc));
    clear(tCrA);

    // if (thread0()) {
    //     print(tAsA);
    //     print(tCrA);
    //     print(tCsA);
    // }
    copy(smem_copy_atom, tCsA, tCrA);

    transform(tCrA, pre_increment{});

    copy(tCrA, tCgc);
}

void test_matrix_copy() {
    using gmem_tiled_copy = decltype(make_tiled_copy(
      Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint32_t>, half_t>{},
      Layout<Shape<_8, _4>>{},
      Layout<Shape<_1, _2>>{}));

    using smem_layout = Layout<Shape<_8, Shape<_2, _4>>, Stride<_2, Stride<_1, _16>>>;
    // using smem_layout = Layout<Shape<_8, _8>>;
    using smem_copy_atom = decltype(make_tiled_copy(
      Copy_Atom<SM75_U32x1_LDSM_N, half_t>{}, Layout<Shape<_8, _4>>{}, Layout<Shape<_1, _2>>{}));

    auto in =
      at::arange(
        decltype(size(gmem_tiled_copy::TiledShape_MN{}))::value,
        at::TensorOptions().dtype(at::kHalf))
        .reshape(
          {size<0>(gmem_tiled_copy::TiledShape_MN{}), size<1>(gmem_tiled_copy::TiledShape_MN{})});

    auto out = at::zeros_like(in);

    half_t *d_in, *d_out;
    hipMalloc((void **)&d_in, in.numel() * in.element_size());
    hipMalloc((void **)&d_out, out.numel() * out.element_size());

    hipMemcpy(d_in, in.data_ptr(), in.numel() * in.element_size(), hipMemcpyHostToDevice);
    hipMemcpy(d_out, out.data_ptr(), out.numel() * out.element_size(), hipMemcpyHostToDevice);

    matrix_copy_kernel<gmem_tiled_copy, smem_copy_atom, smem_layout><<<1, 32>>>(d_in, d_out);
    hipMemcpy(out.data_ptr(), d_out, out.numel() * out.element_size(), hipMemcpyDeviceToHost);
    std::cout << out << std::endl;

    hipFree(d_in);
    hipFree(d_out);
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//        GS --> Async SR--> Ldmatrix
// 1. A as the row major
// 2. A as the col major
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <
  typename OT,
  typename GA_Layout,
  typename SA_Layout,
  typename GS_Tiled_copy,
  typename SR_tiled_copy,
  typename Tiled_MMA_>
__global__ void test_gs_async_sr_ldmatrix_kernel(
  OT const *A,
  OT *out,
  uint32_t *toShareAddr,
  uint32_t *fromShareAddr,
  GA_Layout gA_layout,
  SA_Layout sA_layout,
  GS_Tiled_copy gs_tiled_copy,
  SR_tiled_copy sr_tiled_copy,
  Tiled_MMA_ tiled_mma) {

    __shared__ OT smem_A[cosize_v<SA_Layout>];

    auto gA = make_tensor(make_gmem_ptr(A), gA_layout);
    auto gOut = make_tensor(make_gmem_ptr(out), gA_layout);
    auto sA = make_tensor(make_smem_ptr(smem_A), sA_layout);

    auto sToShareAddr = make_tensor(make_gmem_ptr(toShareAddr), sA_layout);
    auto sFromShareAddr = make_tensor(make_gmem_ptr(fromShareAddr), sA_layout);

    auto gs_thr_copy = gs_tiled_copy.get_thread_slice(threadIdx.x);
    auto tAgA = gs_thr_copy.partition_S(gA);
    auto tAsA = gs_thr_copy.partition_D(sA);
    auto tAgOut = gs_thr_copy.partition_S(gOut);

    auto tAsTo = gs_thr_copy.partition_D(sToShareAddr);
    for (int i = 0; i < size(tAsA); ++i) {
        tAsTo(i) = cast_smem_ptr_to_uint(&tAsA(i));
    }

    copy(gs_tiled_copy, tAgA, tAsA);
    cp_async_fence();
    cp_async_wait<0>();

    auto thr_mma = tiled_mma.get_thread_slice(threadIdx.x);
    auto tCrA = thr_mma.partition_fragment_A(sA);

    auto sr_thr_copy = sr_tiled_copy.get_thread_slice(threadIdx.x);
    auto tCsA = sr_thr_copy.partition_S(sA);
    auto tCrA_view = sr_thr_copy.retile_D(tCrA);

    auto tCsFrom = sr_thr_copy.partition_S(sFromShareAddr);
    for (int i = 0; i < size(tCsA); ++i) {
        tCsFrom(i) = cast_smem_ptr_to_uint(&tCsA(i));
    }

    auto ptr1 = &(tCrA_view(0));
    auto ptr2 = &(tCsA(0));
    copy(sr_tiled_copy, tCsA, tCrA_view);
    transform(tCrA_view, pre_increment{});
    copy(tCrA_view, tAgOut);
}

template <
  typename GS_WT = uint128_t,
  typename OT = half_t,
  typename GA_Layout,
  typename SA_Layout,
  typename GS_ThrLayout,
  typename GS_ValLayout,
  typename MMA_ATOM_OP = SM80_16x8x16_F16F16F16F16_TN,
  typename SR_CP_OP = SM75_U32x4_LDSM_N>
void test_gs_async_sr_ldmatrix_host(
  std::string test_name,
  OT const *A,
  OT *gOut,
  uint32_t *toShareAddr,
  uint32_t *fromShareAddr,
  GA_Layout,
  SA_Layout,
  GS_ThrLayout,
  GS_ValLayout,
  MMA_ATOM_OP,
  SR_CP_OP) {
    auto gA_layout = GA_Layout{};
    auto sA_layout = SA_Layout{};
    auto gs_tiled_copy = make_tiled_copy(
      Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<GS_WT>, OT>{}, GS_ThrLayout{}, GS_ValLayout{});

    auto tiled_mma = TiledMMA<MMA_Atom<MMA_ATOM_OP>>{};
    auto sr_tiled_copy = make_tiled_copy_A(Copy_Atom<SR_CP_OP, OT>{}, tiled_mma);

    if (0) {
        print_latex_header();
        // clang-format off
        print("%%  GA_LAYOUT     : ");print_latex(gA_layout     ,(test_name+"_GA_LAYOUT"    ).c_str());print("\n");
        print("%%  SA_LAYOUT     : ");print_latex(sA_layout     ,(test_name+"_SA_LAYOUT"    ).c_str());print("\n");
        print("%%  GS_TILED_COPY : ");print_latex(gs_tiled_copy ,(test_name+"_GS_TILED_COPY").c_str());print("\n");
        print("%%  TILED_MMA     : ");print_latex(tiled_mma     ,(test_name+"_TILED_MMA"    ).c_str());print("\n");
        print("%%  SR_TILED_COPY : ");print_latex(sr_tiled_copy ,(test_name+"_SR_TILED_COPY").c_str());print("\n");
        // clang-format on

        // copy-gs
        auto [gsA_src_MN, gsA_src_MN_thr] = gs_tiled_copy.get_layoutS_MN();
        auto gsA_src_TV = gs_tiled_copy.get_layoutS_TV();
        auto [gsA_dst_MN, gsA_dst_MN_thr] = gs_tiled_copy.get_layoutD_MN();
        auto gsA_dst_TV = gs_tiled_copy.get_layoutD_TV();

        print_latex(gsA_src_MN, (test_name + "_gsA_src_MN").c_str());
        print_latex(gsA_src_TV, (test_name + "_gsA_src_TV").c_str());
        print_latex(gsA_dst_MN, (test_name + "_gsA_dst_MN").c_str());
        print_latex(gsA_dst_TV, (test_name + "_gsA_dst_TV").c_str());

        // copy-sr
        auto [srA_src_MN, srA_src_MN_thr] = sr_tiled_copy.get_layoutS_MN();
        auto srA_src_TV = sr_tiled_copy.get_layoutS_TV();
        auto [srA_dst_MN, srA_dst_MN_thr] = sr_tiled_copy.get_layoutD_MN();
        auto srA_dst_TV = sr_tiled_copy.get_layoutD_TV();

        print_latex(srA_src_MN, (test_name + "_srA_src_MN").c_str());
        print_latex(srA_src_TV, (test_name + "_srA_src_TV").c_str());
        print_latex(srA_dst_MN, (test_name + "_srA_dst_MN").c_str());
        print_latex(srA_dst_TV, (test_name + "_srA_dst_TV").c_str());

        print_latex_footer();
    }
    // clang-foramt on

    // kernel
    test_gs_async_sr_ldmatrix_kernel<<<1, 32>>>(
      A,
      gOut,
      toShareAddr,
      fromShareAddr,
      gA_layout,
      sA_layout,
      gs_tiled_copy,
      sr_tiled_copy,
      tiled_mma);
}

void test_gs_async_sr_ldmatrix_examples() {
    // test 1 --> row major
    // {
    //     auto gA_layout = Layout<Shape<_16, _16>>{};
    //     auto sA_layout = Layout<Shape<_16, _16>>{};
    //     auto thr_layout = Layout<Shape<_2, _16>>{};
    //     auto val_layout = Layout<Shape<_8, _1>>{};
    //     auto mma_atom_op = SM80_16x8x16_F16F16F16F16_TN{};
    //     auto sr_cp_op = SM75_U32x4_LDSM_N{};

    //     auto h_A = at::arange(
    //                  decltype(size<0>(gA_layout) * size<1>(gA_layout))::value,
    //                  at::TensorOptions().dtype(at::kHalf))
    //                  .reshape({size<0>(gA_layout), size<1>(gA_layout)});
    //     half_t *d_A;
    //     hipMalloc((void **)&d_A, h_A.numel() * h_A.element_size());
    //     hipMemcpy(d_A, h_A.data_ptr(), h_A.numel() * h_A.element_size(),
    //     hipMemcpyHostToDevice);

    //     test_gs_async_sr_ldmatrix_host(
    //       "gs_async_sr_ldmatrix_row",
    //       d_A,
    //       gA_layout,
    //       sA_layout,
    //       thr_layout,
    //       val_layout,
    //       mma_atom_op,
    //       sr_cp_op);
    // }

    // test 2 --> col major
    {
        auto gA_layout = Layout<Shape<_16, _16>, Stride<_16, _1>>{};
        auto sA_layout = Layout<Shape<_16, _16>, Stride<_16, _1>>{};
        auto thr_layout = Layout<Shape<_16, _2>>{};
        auto val_layout = Layout<Shape<_1, _8>>{};
        auto mma_atom_op = SM80_16x8x16_F16F16F16F16_TN{};
        auto sr_cp_op = SM75_U32x4_LDSM_N{};

        auto h_A = at::arange(
                     decltype(size<0>(gA_layout) * size<1>(gA_layout))::value,
                     at::TensorOptions().dtype(at::kHalf))
                     .reshape({size<0>(gA_layout), size<1>(gA_layout)});
        auto h_out = at::zeros_like(h_A);

        auto h_toShareAddr = at::arange(
                               decltype(size<0>(gA_layout) * size<1>(gA_layout))::value,
                               at::TensorOptions().dtype(at::kInt))
                               .reshape({size<0>(gA_layout), size<1>(gA_layout)});
        auto h_fromShareAddr = at::arange(
                                 decltype(size<0>(gA_layout) * size<1>(gA_layout))::value,
                                 at::TensorOptions().dtype(at::kInt))
                                 .reshape({size<0>(gA_layout), size<1>(gA_layout)});

        half_t *d_A, *d_out;
        uint32_t *d_toShareAddr, *d_fromShareAddr;
        hipMalloc((void **)&d_A, h_A.numel() * h_A.element_size());
        hipMalloc((void **)&d_out, h_out.numel() * h_out.element_size());
        hipMalloc((void **)&d_toShareAddr, h_toShareAddr.numel() * h_toShareAddr.element_size());
        hipMalloc(
          (void **)&d_fromShareAddr, h_fromShareAddr.numel() * h_fromShareAddr.element_size());
        hipMemcpy(d_A, h_A.data_ptr(), h_A.numel() * h_A.element_size(), hipMemcpyHostToDevice);

        test_gs_async_sr_ldmatrix_host(
          "gs_async_sr_ldmatrix_col",
          d_A,
          d_out,
          d_toShareAddr,
          d_fromShareAddr,
          gA_layout,
          sA_layout,
          thr_layout,
          val_layout,
          mma_atom_op,
          sr_cp_op);

        hipMemcpy(
          h_out.data_ptr(), d_out, h_A.numel() * h_A.element_size(), hipMemcpyDeviceToHost);
        hipMemcpy(
          h_toShareAddr.data_ptr(), d_toShareAddr, h_toShareAddr.numel() * h_toShareAddr.element_size(), hipMemcpyDeviceToHost);
        hipMemcpy(
          h_fromShareAddr.data_ptr(), d_fromShareAddr, h_fromShareAddr.numel() * h_fromShareAddr.element_size(), hipMemcpyDeviceToHost);
        std::cout << h_out << std::endl;
        std::cout << h_toShareAddr << std::endl;
        std::cout << h_fromShareAddr << std::endl;
    }
}

int main() {
    // test_normal_copy();

    // test_matrix_copy();
    // test_copy_host();

    test_gs_async_sr_ldmatrix_examples();

    hipDeviceReset();
}