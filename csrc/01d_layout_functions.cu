#include "hip/hip_runtime.h"
#include "cute/tensor.hpp"
#include "cute/algorithm/tuple_algorithms.hpp"
#include "cute/arch/mma_sm80.hpp"
#include "cute/atom/mma_atom.hpp"
#include "cute/container/tuple.hpp"
#include "cute/int_tuple.hpp"
#include "cute/numeric/int.hpp"
#include "cute/numeric/integral_constant.hpp"
#include "cute/util/print.hpp"
#include "latex.hpp"
#include <cute/layout.hpp>
#include <cute/algorithm/copy.hpp>
#include <string>

using namespace cute;

template <typename Args>
void custom_print(Args args, int ps = -1) {
    switch (ps) {
        case 0:
            print_layout(args);
            break;
        case 1:
            print_latex(args);
            break;
        default:
            print(args);
            break;
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                              Transform Leaf
////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Tuple, typename Fn>
void test_transform_leaf() {
    auto res = transform_leaf(Tuple{}, Fn{});

    // clang-format off
    print("Input  : ");print(Tuple{});print("\n");
    print("Output : ");print(res);print("\n");
    // clang-format on
}

void test_transform_leaf_examples() {
    test_transform_leaf<tuple<_2, _m1, tuple<_m5, _1>>, abs_fn>();
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                              Transform
////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename TupleA, typename TupleB, typename Fn>
void test_transform() {
    auto res = transform(TupleA{}, TupleB{}, Fn{});

    // clang-format off
    print("Input  : ");print(TupleA{});print(" , ");print(TupleB{});print("\n");
    print("Output : ");print(res);print("\n");
    // clang-format on
}

void test_transform_examples() {
    {
        print("Addition : \n");
        test_transform<tuple<_2, _10>, tuple<_5, _6>, plus>();
    }
    {
        print("Max  : \n");
        test_transform<tuple<_2, _10>, tuple<_5, _6>, max_fn>();
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                              Find
////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T, typename X>
void test_find() {
    auto res = find(T{}, X{});

    // clang-format off
    print("Input  : ");print(T{});print("\n");
    print("Output : ");print(res);print("\n");
    // clang-format on
}

void test_find_examples() {
    test_find<tuple<_1, _4, _5, _2, _6>, _4>();
    test_find<tuple<_1, _4, _5, _2, _6>, _5>();
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                              Find if
////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T, typename F>
void test_find_if(T t, F &&f) {
    auto res = find_if(T{}, f);

    // clang-format off
    print("Input  : ");print(T{});print("\n");
    print("Output : ");print(res);print("\n");
    // clang-format on
}

void test_find_if_examples() {
    {
        print("Find the Value which greater than 5 and return index : \n");
        test_find_if(tuple<_1, _2, _5, _7>{}, [&](auto const &i) { return greater{}(i, _5{}); });
    }
    {
        print("Find the Value which equal to 5 and return index : \n");
        test_find_if(tuple<_1, _2, _5, _7>{}, [&](auto const &i) { return equal_to{}(i, _5{}); });
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                              Compact Col Major
////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Shape>
void test_compact_col_major() {
    auto res = compact_col_major(Shape{});
    print("Input  : ");
    print(Shape{});
    print("\n");
    print("Output : ");
    print(res);
    print("\n");
}

void test_compact_col_major_examples() {
    test_compact_col_major<Shape<_4>>();
    test_compact_col_major<Shape<_4, _2>>();
    test_compact_col_major<Shape<_4, _2, _8>>();
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                              Inverse seq
////////////////////////////////////////////////////////////////////////////////////////////////////

template <int Start, typename Shape, typename Stride>
void test_inverse_seq() {
    auto res = detail::inverse_seq<Start>(Shape{}, Stride{}, seq<>{});

    // clang-format off
    print("Input  : ");print(Shape{});print(Stride{});print("\n");
    print("Output : ");print(res);print("\n");
    // clang-format on
}

void test_inverse_seq_examples() {
    test_inverse_seq<1, tuple<_4>, tuple<_1>>();
    test_inverse_seq<1, tuple<_4, _4>, tuple<_1, _4>>();
    test_inverse_seq<1, tuple<_4, _4>, tuple<_1, _5>>();
    test_inverse_seq<1, tuple<_4, _5>, tuple<_1, _4>>();
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                              Right Inverse
////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Layout>
void test_right_inverse(int ps) {
    auto res = right_inverse(Layout{});

    // clang-format off
    print("Input  : ");print(Layout{});print("\n");
    print("Output : ");print(res);print("\n");
    // clang-format on
}

void test_right_inverse_examples(int ps) {
    // test_right_inverse<Layout<Shape<_4>, Stride<_1>>>(ps);
    // test_right_inverse<Layout<Shape<_4, _4>, Stride<_1, _4>>>(ps);
    // test_right_inverse<Layout<Shape<_4, _4>, Stride<_1, _5>>>(ps);
    // test_right_inverse<Layout<Shape<_4, _5>, Stride<_1, _4>>>(ps);
    // test_right_inverse<Layout<Shape<_4, _5>, Stride<_5, _1>>>(ps);
    test_right_inverse<Layout<Shape<_16, Shape<_8, _8>>, Stride<_8, Stride<_128, _1>>>>(ps);
    // test_right_inverse<Layout<Shape<Shape<_3,_2>,Shape<_4,_2>>,Stride<Stride<_4,_1>,Stride<_12,_2>>>>(ps);
    // test_right_inverse<Layout<Shape<_32,_8>,Stride<_1,_32>>>(ps);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                              Composition
////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename A, typename B>
void test_composition(int ps) {
    // auto res = composition(A{}, B{});
    auto res = A{}.compose(B{}, _);

    // clang-format off
    print("Input  : ");custom_print(A{},ps);print(" , ");custom_print(B{},ps);print("\n");
    print("Output : ");custom_print(res,ps);print("\n");
    // clang-format on
}

void test_composition_examples(int ps) {

    // test_composition<
    //   Layout<Shape<Int<20>, _2>, Stride<_16, _4>>,
    //   Layout<Shape<_4, _5>, Stride<_1, _4>>>(ps);

    test_composition<Layout<Shape<Shape<_2, _4>, Shape<_3, _5>>>, Layout<Shape<_1, _2>>>(ps);

    // test_composition<
    //   Layout<Shape<_2, _32>, Stride<_32, _1>>,
    //   Layout<Shape<Shape<_8, _4>, _8>, Stride<Stride<_8, _0>, _1>>>(ps);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                Right Inverse of Ref then Compose to Src
////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename LayoutRef, typename LayoutSrc>
void test_RIRCS(int ps) {
    auto res = right_inverse(LayoutRef{}).compose(LayoutSrc{});

    // clang-format off
    print("Reference Layout : ");custom_print(LayoutRef{},ps);print("\n");
    print("Source    Layout : ");custom_print(LayoutSrc{},ps);print("\n");
    print("Result    Layout : ");custom_print(res,ps);print("\n");
    // clang-format on
}

void test_RIRCS_examples(int ps) {
    {
        print("Source : 128b , Val : 16b \n");
        using copy_atom = Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, half_t>;
        print(copy_atom{});
        test_RIRCS<typename copy_atom::ValLayoutRef, typename copy_atom::ValLayoutSrc>(ps);
        print("\n");
    }

    {
        print("Source : 128b , Val : 8b \n");
        using copy_atom = Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, int8_t>;
        print(copy_atom{});
        test_RIRCS<typename copy_atom::ValLayoutRef, typename copy_atom::ValLayoutSrc>(ps);
        print("\n");
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                              Rank
// Number of mode in layout is rank.
////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Layout>
void test_rank() {
    auto res = rank_v<Layout>;
    print("Rank of Layout : ");
    print(Layout{});
    print(" is : ");
    print(res);
    print("\n");
}

void test_rank_examples() {
    test_rank<Layout<Shape<_1>>>();
    test_rank<Layout<Shape<_16, _8>, Stride<_8, _1>>>();
    test_rank<Layout<Shape<_1, _8>>>();
    test_rank<Layout<Shape<Shape<_2, _4>, _2>>>();
    test_rank<Layout<Shape<Shape<_2, _4>, _2, _1>>>();
    test_rank<Layout<Shape<Shape<_2, _4>, _2, _0>>>();
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                              Append
// If N is less than size tuple throw error
// if N is equal to size of tuple return same tuple
// if N is great than size of tuple then append up to N.
////////////////////////////////////////////////////////////////////////////////////////////////////
template <typename T, typename X, int N>
void test_append() {
    auto res = append<N>(T{}, X{});

    // clang-format off
    print("Input  : ");print(T{});print(" , ");print(X{});print("\n");
    print("Output : ");print(res);print("\n");
    // clang-format on
}

void test_append_examples() {
    // {
    //     print("Append at 1st position \n");
    //     test_append<Layout<Shape<_16, _8>, Stride<_8, _1>>,
    //                 Layout<_1>,
    //                 1>();
    //     test_append<Layout<Shape<_1, _8>>,
    //                 Layout<_1>,
    //                 1>();
    // }
    {
        print("Append at 2nd position \n");
        test_append<Layout<Shape<_16, _8>, Stride<_8, _1>>, Layout<_1>, 2>();
        test_append<Layout<Shape<_1, _8>>, Layout<_1>, 2>();
    }

    {
        print("Append at 3rd position \n");
        test_append<Layout<Shape<_16, _8>, Stride<_8, _1>>, Layout<_1>, 3>();
        test_append<Layout<Shape<_1, _8>>, Layout<_1>, 3>();
    }
    {
        print("Append at 4th position \n");
        test_append<Layout<Shape<_16, _8>, Stride<_8, _1>>, Layout<_2>, 4>();
        test_append<Layout<Shape<_1, _8>>, Layout<_2>, 4>();
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                              Repeat
////////////////////////////////////////////////////////////////////////////////////////////////////

template <int N>
void test_repeat() {
    auto res = repeat<N>(_);

    // clang-format off
    print(" Output : ");print(res);print("\n");
    // clang-format on
}

void test_repeat_examples() {
    test_repeat<1>();
    test_repeat<2>();
    test_repeat<3>();
    test_repeat<4>();
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                          Raked product
////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Tile, typename MatOfTiles>
void test_raked_product(int ps) {
    auto res = raked_product(Tile{}, MatOfTiles{});

    // clang-format off
    print("Input  : ");custom_print(Tile{},ps);print(" , ");custom_print(MatOfTiles{},ps);print("\n");
    print("Output : ");custom_print(res,ps);print("\n");
    // clang-format on
}

void test_raked_product_examples(int ps) {
    test_raked_product<Layout<Shape<_16, _8>, Stride<_8, _1>>, Layout<Shape<_1, _8>>>(ps);
    // test_raked_product<Layout<Shape<_2, _2>>, Layout<Shape<_3, _4>>>(ps);
    // test_raked_product<Layout<Shape<_32, _1>>, Layout<Shape<_1, _8>>>(ps);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                          With Shape
////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Layout, typename Shape>
void test_with_shape(int ps) {
    auto res = Layout{}.with_shape(Shape{});

    // clang-format off
    print("Input  : ");print(Layout{});print(" , ");print(Shape{});print("\n");
    print("Output : ");custom_print(res,ps);print("\n");
    // clang-format on
}

void test_with_shape_examples(int ps) {
    test_with_shape<Layout<Shape<_8, _128>, Stride<_128, _1>>, Shape<_128, _8>>(ps);
    // test_with_shape<Layout<Shape<_2,_2,_3,_4>,Stride<_3,_24,_1,_6>>,Shape<_4,_12>>(ps);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                          zipped product
////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Layout, typename Tile>
void test_zipped_product(int ps) {
    auto res = zipped_product(Layout{}, Tile{});

    // clang-format off
    print("Input : ");print(Layout{});print(" , ");print(Tile{});print("\n");
    custom_print(res,ps);print("\n");
    // clang-format on
}

void test_zipped_product_examples(int ps) {
    test_zipped_product<Layout<Shape<_2, _2>>, Layout<Shape<_2, _3>>>(ps);
    test_zipped_product<
      Layout<Shape<_2, _3>, Stride<_3, _1>>,
      Layout<Shape<_2, _2>, Stride<_2, _1>>>(ps);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                          zipped divide
// Layout   ==> thr,val
// tile     ==> g_thr,g_val  gather
// zip_div  ==> (g_thr,g_val),(r_thr,r_val) reminder

// Layout   ==> ((_2,_3),(_4,_6)):((_1,_2),(_6,_24))
// tile     ==> (_1,_8)
// zip_div  ==> ((_1,_8),(_6,_3)):((_0,_6),(_1,_48))
////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Layout, typename Tile>
void test_zipped_divide(int ps) {
    auto res = zipped_divide(Layout{}, Tile{});

    // clang-format off
    print("Input : ");custom_print(Layout{},ps);print(" , ");print(Tile{});print("\n");
    custom_print(res,ps);print("\n");
    // clang-format on
}

void test_zipped_divide_examples(int ps) {
    test_zipped_divide<Layout<Shape<Shape<_2, _3>, Shape<_4, _6>>>, Shape<_1, _8>>(ps);
    test_zipped_divide<
      Layout<Shape<Shape<_2, _3>, Shape<_4, _6>>, Stride<Stride<_4, Int<48>>, Stride<_1, _8>>>,
      Shape<_1, _8>>(ps);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                          tiled product
////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Layout, typename Tile>
void test_tiled_product() {
    auto res    = tiled_product(Layout{}, Tile{});
    auto tensor = make_counting_tensor(res);

    // clang-format off
    print("Input : ");print(Layout{});print(" , ");print(Tile{});print("\n");
    print_tensor(tensor);print("\n\n");
    // clang-format on
}

void test_tiled_product_examples() {
    test_tiled_product<Layout<Shape<_2, _2>>, Layout<Shape<_2, _3>>>();
    test_tiled_product<
      Layout<Shape<_2, _3>, Stride<_3, _1>>,
      Layout<Shape<_2, _2>, Stride<_2, _1>>>();
    test_tiled_product<Layout<_32>, Layout<Shape<_2, _2, _1>, Stride<_1, _2, _0>>>();
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//  *                      max common vector
//  ? Return Int<N> such that N is the maximum number of contiguous elements
//  ? that logically correspond in the layouts of @a a and @a b. This is,
//  ? the number of elements that could reasonably be "vectorized" in the layouts.

// Examples
// Input  : (_4,_4):(_1,_4) , (_2,_2):(_1,_2)
// Output : _4

// Input  : (_2,_2):(_1,_2) , (_4,_4):(_1,_4)
// Output : _16

// Input  : (_4,_4):(_4,_1) , (_2,_2):(_2,_1)
// Output : _1

// Input  : (_4,_4):(_1,_8) , (_2,_2):(_1,_4)
// Output : _2

// Input  : _4:_2 , _4:_2
// Output : _1
// Even shape and stride are same this not contiguous in memory layout
////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename SrcLayout, typename DstLayout>
void test_max_common_vector() {
    auto res = max_common_vector(SrcLayout{}, DstLayout{});

    // clang-format off
    print("Input  : ");print(SrcLayout{});print(" , ");print(DstLayout{});print("\n");
    print("Output : ");print(res);print("\n\n");
    // clang-format on
}

void test_max_common_vector_examples() {
    // col major
    test_max_common_vector<Layout<Shape<_4, _4>>, Layout<Shape<_2, _2>>>();
    test_max_common_vector<Layout<Shape<_2, _2>>, Layout<Shape<_4, _4>>>();

    // row  major
    test_max_common_vector<
      Layout<Shape<_4, _4>, Stride<_4, _1>>,
      Layout<Shape<_2, _2>, Stride<_2, _1>>>();
    test_max_common_vector<
      Layout<Shape<_2, _2>, Stride<_2, _1>>,
      Layout<Shape<_4, _4>, Stride<_4, _1>>>();

    // row and col major
    test_max_common_vector<
      Layout<Shape<_4, _4>, Stride<_4, _1>>,
      Layout<Shape<_2, _2>, Stride<_1, _2>>>();
    test_max_common_vector<
      Layout<Shape<_2, _2>, Stride<_2, _1>>,
      Layout<Shape<_4, _4>, Stride<_1, _4>>>();

    // stride
    test_max_common_vector<
      Layout<Shape<_4, _4>, Stride<_1, _8>>,
      Layout<Shape<_2, _2>, Stride<_1, _4>>>();
    test_max_common_vector<
      Layout<Shape<_2, _2>, Stride<_1, _4>>,
      Layout<Shape<_4, _4>, Stride<_1, _8>>>();

    test_max_common_vector<Layout<_4, _2>, Layout<_4, _2>>();

    test_max_common_vector<
      Layout<Shape<Shape<_8, _1>, _1, _1>, Stride<Stride<_1, _0>, _0, _0>>,
      Layout<Shape<Shape<_8, _1>, _1, _1>, Stride<Stride<_1, _0>, _0, _0>>>();
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                              Thread Layout and Val Layout
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename ThrLayout, typename ValLayout>
void test_build_layoutTV(std::string test_name) {
    auto interleaved    = raked_product(ThrLayout{}, ValLayout{});
    auto val_thr_layout = right_inverse(interleaved);
    auto thr_val_layout =
      val_thr_layout.with_shape(make_shape(size(ThrLayout{}), size(ValLayout{})));
    auto mn_layout = make_layout(product_each(shape(interleaved)));
    auto zip_div   = zipped_divide(mn_layout, shape(ValLayout{}));

    print_latex(mn_layout, (test_name + std::string("_mn")).c_str());
    print_latex(zip_div, (test_name + std::string("_zip_div")).c_str());
    print_latex(interleaved, (test_name + std::string("_interleaved")).c_str());
    // clang-format off
    print("%% val_thr_layout : ");print(val_thr_layout);
    // clang-format on
    print_latex(thr_val_layout, (test_name + std::string("_thr_val_layout")).c_str());
}

void test_build_layoutTV_examples() {
    print_latex_header();
    test_build_layoutTV<Layout<Shape<_2, _3>>, Layout<Shape<_4, _5>>>("T2x3_V4x5");
    print_latex_footer();
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                                  TidFrag
//////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename ThrLayout, typename ValLayout>
void test_tidFrag(std::string test_name) {
    using tiled_copy = decltype(make_tiled_copy(
      Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, half_t>{}, ThrLayout{}, ValLayout{}));

    auto mn_layout =
      make_layout(product_each(zip((typename tiled_copy::TiledShape_MN{}), make_tuple(2, 2))));

    auto res        = tiled_copy::tidfrg_S(mn_layout);
    auto res_tensor = make_counting_tensor(res);

    // print_latex(mn_layout, (test_name + "_mn_layout").c_str());
    // print_latex(res, (test_name + "_tidfrg_S").c_str());
    print(mn_layout);
    print("\n");
    print(res);
    print("\n");
    print(coalesce(res));
    print_tensor(res_tensor);
}

void test_tidFrag_examples() {
    // print_latex_header();
    test_tidFrag<Layout<Shape<_2, _3>>, Layout<Shape<_4, _6>>>("tidFrag_T2x3_V4x6");
    // print_latex_footer();
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                                  Tile to Thread Frag
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
template <typename CopyAtom, typename ThrLayout, typename ValLayout, typename RestShape_MN>
void test_tile_thrFrag(std::string test_name, int ps) {
    using tiled_copy = decltype(make_tiled_copy(CopyAtom{}, ThrLayout{}, ValLayout{}));

    auto thr_copy = tiled_copy{}.get_thread_slice(1);

    auto mn_layout =
      make_layout(product_each(zip((typename tiled_copy::TiledShape_MN{}), RestShape_MN{})));

    auto mn_tensor = make_counting_tensor(mn_layout);

    // auto mn_layout = make_layout(typename tiled_copy::TiledShape_MN{});

    auto tile    = zipped_divide(mn_layout, typename tiled_copy::Tiler_MN{});
    auto ref2trg = right_inverse(typename tiled_copy::AtomLayoutRef{})
                     .compose(typename tiled_copy::AtomLayoutDst{});

    auto atom_layout_TV = zipped_divide(
      typename tiled_copy::TiledLayout_TV{},
      make_shape(typename tiled_copy::AtomNumThr{}, typename tiled_copy::AtomNumVal{}));

    auto trg_layout_TV = atom_layout_TV.compose(ref2trg, _);
    auto thrval2mn     = coalesce(zip(trg_layout_TV), Shape<_1, Shape<_1, _1>>{});
    auto tv_tensor     = tile.compose(thrval2mn, _);
    auto res           = tv_tensor(make_coord(_, _), _);
    auto res_tensor    = make_counting_tensor(res);
    auto part          = thr_copy.partition_S(mn_tensor);

    // clang-format off
    print("%%  AtomThrID      : ");print       (typename tiled_copy::AtomThrID      {}                                         );print("\n");
    print("%%  AtomLayoutSrc  : ");custom_print(typename tiled_copy::AtomLayoutSrc  {},(test_name+"_AtomLayoutSrc").c_str(),ps );print("\n");
    print("%%  AtomLayoutDst  : ");custom_print(typename tiled_copy::AtomLayoutDst  {},(test_name+"_AtomLayoutDst").c_str(),ps );print("\n");
    print("%%  AtomLayoutRef  : ");custom_print(typename tiled_copy::AtomLayoutRef  {},(test_name+"_AtomLayoutRef").c_str(),ps );print("\n");
    print("%%  AtomNumThr     : ");print       (typename tiled_copy::AtomNumThr     {}                                         );print("\n");
    print("%%  AtomNumVal     : ");print       (typename tiled_copy::AtomNumVal     {}                                         );print("\n");
    print("%%  Tiler_MN       : ");print       (typename tiled_copy::Tiler_MN       {}                                         );print("\n");
    print("%%  TiledShape_MN  : ");print       (typename tiled_copy::TiledShape_MN  {}                                         );print("\n");
    print("%%  TiledLayout_TV : ");custom_print(typename tiled_copy::TiledLayout_TV {},(test_name+"_TiledLayout_TV").c_str(),ps);print("\n");
    print("%%  TiledNumThr    : ");print       (typename tiled_copy::TiledNumThr    {}                                         );print("\n");
    print("%%  TiledNumVal    : ");print       (typename tiled_copy::TiledNumVal    {}                                         );print("\n");
    if(ps==1)
        {print("%%  TiledCopy      : ");print_latex(         tiled_copy                 {},(test_name+"_TiledCopy").c_str()     );print("\n");}


    print("%%  Mn_Layout      : ");custom_print(mn_layout      ,(test_name+"_Mn_Layout"      ).c_str(),ps );print("\n");
    print("%%  Tile           : ");custom_print(tile           ,(test_name+"_Tile"           ).c_str(),ps );print("\n");
    print("%%  Ref2trg        : ");custom_print(ref2trg        ,(test_name+"_Ref2trg"        ).c_str(),ps );print("\n");
    print("%%  Atom_Layout_Tv : ");custom_print(atom_layout_TV ,(test_name+"_Atom_Layout_Tv" ).c_str(),ps );print("\n");
    print("%%  Trg_Layout_Tv  : ");custom_print(trg_layout_TV  ,(test_name+"_Trg_Layout_Tv"  ).c_str(),ps );print("\n");
    print("%%  Thrval2mn      : ");custom_print(thrval2mn      ,(test_name+"_Thrval2mn"      ).c_str(),ps );print("\n");
    print("%%  Tv_Tensor      : ");custom_print(tv_tensor      ,(test_name+"_Tv_Tensor"      ).c_str(),ps );print("\n");
    
    print("%%  Tile2Frag      : " );print       (res        );print("\n");
    if(ps !=1)
        {
        print("%%  TiledCopy      : " );print       ( tiled_copy {} );print("\n");
        print("%% Tile2Frag       : " );print_tensor( res_tensor    );print("\n");
        print("%% Partition       : " );print_tensor( part          );print("\n");
        }
    // clang-format on
}

void test_tile_thrFrag_examples(int ps) {
    if (ps == 1)
        print_latex_header();
    // test_tile_thrFrag<
    //   Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint32_t>, half_t>,
    //   Layout<Shape<_2, _2>>,
    //   Layout<Shape<_2, _2>>,
    //   Shape<_2, _1>>("tile2Frag_T2x2_V2x2", ps);
    // test_tile_thrFrag<
    //   Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, half_t>,
    //   Layout<Shape<_2, _3>>,
    //   Layout<Shape<_4, _6>>,
    //   Shape<_2, _2>>("tile2Frag_T2x3_V4x6", ps);
    test_tile_thrFrag<
      Copy_Atom<SM75_U32x1_LDSM_N, half_t>,
      Layout<Shape<_4, _8>>,
      Layout<Shape<_2, _1>>,
      Shape<_2, _2>>("tile2Frag_LDSM_T4x8_V4x2", ps);
    if (ps == 1)
        print_latex_footer();
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                              MMa thr Frag
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <
  typename Operation,
  typename AtomLayoutMNK   = Layout<Shape<_1, _1, _1>>,
  typename ValLayoutMNK    = Layout<Shape<_1, _1, _1>>,
  typename PermutationsMNK = Tile<Underscore, Underscore, Underscore>>
void test_mma_thr_Frag(std::string test_name, int ps) {
    using tiled_mma = TiledMMA<MMA_Atom<Operation>, AtomLayoutMNK, ValLayoutMNK, PermutationsMNK>;

    // clang-format off
    print("%%  AtomShape_MNK   : ");print       (typename tiled_mma::AtomShape_MNK  {}                                         );print("\n");
    print("%%  AtomLayoutC_TV  : ");custom_print(typename tiled_mma::AtomLayoutC_TV {},(test_name+"_AtomLayoutC_TV").c_str(),ps);print("\n");
    print("%%  AtomLayoutA_TV  : ");custom_print(typename tiled_mma::AtomLayoutA_TV {},(test_name+"_AtomLayoutA_TV").c_str(),ps);print("\n");
    print("%%  AtomLayoutB_TV  : ");custom_print(typename tiled_mma::AtomLayoutB_TV {},(test_name+"_AtomLayoutB_TV").c_str(),ps);print("\n");
    print("%%  AtomThrID       : ");print       (typename tiled_mma::AtomThrID      {}                                         );print("\n");
    print("%%  TiledShape_MNK  : ");print       (typename tiled_mma::TiledShape_MNK {}                                         );print("\n");
    print("%%  ThrLayoutVMNK   : ");print       (typename tiled_mma::ThrLayoutVMNK  {}                                         );print("\n");
    print("%%  TidLayout       : ");print       (typename tiled_mma::TidLayout      {}                                         );print("\n");
    // clang-format on

    auto ref = make_layout(make_shape(
      size<0>(typename tiled_mma::TiledShape_MNK{}),
      size<1>(typename tiled_mma::TiledShape_MNK{})));
    auto t_tile =
      make_tile(left_inverse(get<0>(PermutationsMNK{})), left_inverse(get<1>(PermutationsMNK{})));
    auto t_tensor = logical_divide(ref, t_tile);
    auto a_tile   = make_tile(
      make_layout(size<0>(typename tiled_mma::AtomShape_MNK{})),
      make_layout(size<1>(typename tiled_mma::AtomShape_MNK{})));
    auto a_tensor  = zipped_divide(t_tensor, a_tile);
    auto tv_tensor = a_tensor.compose(typename tiled_mma::AtomLayoutC_TV{}, _);
    auto thr_tile  = make_tile(
      _,
      make_tile(
        make_layout(size<1>(typename tiled_mma::ThrLayoutVMNK{})),
        make_layout(size<2>(typename tiled_mma::ThrLayoutVMNK{}))));
    auto thr_tensor = zipped_divide(tv_tensor, thr_tile);
    auto tid_frag   = thr_tensor.compose(typename tiled_mma::TidLayout{}, _);

    // clang-format off
    print("%%  Ref        : ");custom_print (ref        ,(test_name+"_Ref").c_str(),ps       );print("\n");
    print("%%  T_Tile     : ");print        (t_tile                                          );print("\n");
    print("%%  T_Tensor   : ");custom_print (t_tensor   ,(test_name+"_T_Tensor").c_str(),ps  );print("\n");
    print("%%  A_Tile     : ");print        (a_tile                                          );print("\n");
    print("%%  A_Tensor   : ");custom_print (a_tensor   ,(test_name+"_A_Tensor").c_str(),ps  );print("\n");
    print("%%  Tv_Tensor  : ");custom_print (tv_tensor  ,(test_name+"_Tv_Tensor").c_str(),ps );print("\n");
    print("%%  Thr_Tile   : ");print        (thr_tile                                        );print("\n");
    print("%%  Thr_Tensor : ");custom_print (thr_tensor,(test_name+"_Thr_Tensor").c_str(),ps );print("\n");
    print("%%  Tid_Frag   : ");custom_print (tid_frag   ,(test_name+"_Tid_Frag").c_str(),ps  );print("\n");
    // clang-format on
}

void test_mma_thr_Frag_examples(int ps) {
    if (ps == 1)
        print_latex_header();
    test_mma_thr_Frag<SM80_16x8x16_F16F16F16F16_TN>("mma_m16n8k16_f16f16f16f16", ps);
    if (ps == 1)
        print_latex_footer();
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                                  Data and Thread Arrangement for mma
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template <typename MMA_Atom_OP_, typename Atom_Layout, typename Val_Layout>
void test_data_and_thread_arrangement_for_mma(std::string test_name) {
    auto tiled_mma = TiledMMA<MMA_Atom<MMA_Atom_OP_>, Atom_Layout, Val_Layout>{};
    print_latex(tiled_mma, ("TiledMMA_" + test_name).c_str());
}

void test_data_and_thread_arrangement_for_mma_examples()
{
    print_latex_header();
    test_data_and_thread_arrangement_for_mma<SM80_16x8x8_F32TF32TF32F32_TN,Layout<Shape<_1,_1,_1>>,Layout<Shape<_1,_1,_1>>>("A1x1x1_V1x1x1");
    test_data_and_thread_arrangement_for_mma<SM80_16x8x8_F32TF32TF32F32_TN,Layout<Shape<_2,_2,_1>>,Layout<Shape<_1,_1,_1>>>("A2x2x1_V1x1x1");
    test_data_and_thread_arrangement_for_mma<SM80_16x8x8_F32TF32TF32F32_TN,Layout<Shape<_2,_2,_1>>,Layout<Shape<_1,_2,_1>>>("A2x2x1_V1x2x1");
    print_latex_footer();
}

int main(int argc, char *argv[]) {
    // print_select
    [[maybe_unused]] int ps{-1};
    if (argc >= 2)
        ps = atoi(argv[1]);

    // test_transform_leaf_examples();
    // test_transform_examples();
    // test_find_examples();
    // test_find_if_examples();
    // test_compact_col_major_examples();
    // test_inverse_seq_examples();
    // test_right_inverse_examples(ps);
    // test_composition_examples(ps);
    // test_RIRCS_examples(ps);
    // test_repeat_examples();
    // test_rank_examples();
    // test_append_examples();
    // test_raked_product_examples(ps);
    // test_with_shape_examples(ps);
    // test_zipped_product_examples(ps);
    // test_zipped_divide_examples(ps);
    // test_tiled_product_examples();
    // test_max_common_vector_examples();
    // test_build_layoutTV_examples();
    // test_tidFrag_examples();
    // test_tile_thrFrag_examples(ps);
    // test_mma_thr_Frag_examples(ps);
    test_data_and_thread_arrangement_for_mma_examples();
}